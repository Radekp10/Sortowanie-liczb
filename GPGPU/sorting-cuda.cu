#include "hip/hip_runtime.h"
#include <stdio.h> //printf
#include <time.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#define SEED 4567
#define N 10000
#define THREADS_PER_BLOCK 512


//Prototypy funkcji
__global__ void EvenPhase(int* array, int numberOfElements);
__global__ void OddPhase(int* array, int numberOfElements);
int * TranspositionSort(int* array, int numberOfElements, double* time);
int GetRandomInt(int lowerBound, int upperBound);
int * GenerateEvenSizeRandomElementArray(int seed, int* numberOfElements,
    int lowerBound, int upperBound);
int * CopyArray(int* array, int numberOfElements);
void PrintElementArray(int* array, int numberOfElements);


__global__ void EvenPhase(int* array, int numberOfElements)
{
    //Globalny ID watku
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    //Zmienne pomocnicza
    int tmp = 0;
    int upperBound = (int)numberOfElements / 2;
    //Sprawdzenie czy nie wychodzimy poza granice
    if (id < upperBound)
    {
        //Por�wnujemy pary element�w (x0,x1), (x2, x3), ...
        //Je�li x[i] > x[i+1] to zamieniamy miejscami
        if (array[2 * id] > array[2 * id + 1])
        {
            tmp = array[2 * id];
            array[2 * id] = array[2 * id + 1];
            array[2 * id + 1] = tmp;
        }
    }
}


__global__ void OddPhase(int* array, int numberOfElements)
{
    //Globalny ID watku
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    //Zmienne pomocnicza
    int tmp = 0;
    int upperBound = ((int)numberOfElements / 2) - 1;
    //Sprawdzenie czy nie wychodzimy poza granice
    if (id < upperBound)
    {
        //Por�wnujemy pary element�w (x1,x2), (x3, x4), ...
        //Je�li x[i] > x[i+1] to zamieniamy miejscami
        if (array[2 * id + 1] > array[2 * id + 2])
        {
            tmp = array[2 * id + 1];
            array[2 * id + 1] = array[2 * id + 2];
            array[2 * id + 2] = tmp;
        }
    }
}


int * TranspositionSort(int* array, int numberOfElements, double* time)
{
    //Czasy rozpoczecia i zakonczenia sortowania
    clock_t start, stop;
    //Sprawdzanie warunk�w pocz�tkowych sortowania
    if (numberOfElements % 2 == 1)
    {
        printf("Uneven number of elements! Array hasn't been sorted.");
        return array;
    }

    //Kopie zmiennych na hoscie
    int* newArray = CopyArray(array, numberOfElements);

    //Kopie zmiennych na device
    int* d_newArray;

    //Wielkosc pamieci do zaalokowania
    int array_size = numberOfElements * sizeof(int);

    //Alokacja pamieci na device dla kopii zmiennych
    hipMalloc(&d_newArray, array_size);

    //Rozmiar gridu
    int gridSizeEvenPhase, gridSizeOddPhase;

    // Liczba blokow w gridzie (dzielimy przez 2, bo elemnety badane sa parami)
    gridSizeEvenPhase = (int)numberOfElements / 2;
    gridSizeOddPhase = (int)numberOfElements / 2 - 1; //Zerowy i ostatni element tablicy nie maja pary, wiec liczba iteracji jest o 1 mniejsza

    //Liczba blokow
    //Zabezpieczenie, by nie bylo za malo blokow (gdy numberOfElements nie dzieli sie na NUM_THREADS)
    int numBlocksEvenPhase = (gridSizeEvenPhase + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int numBlocksOddPhase = (gridSizeOddPhase + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    //Fazy sortowania
    start = clock();
    hipMemcpy(d_newArray, newArray, array_size, hipMemcpyHostToDevice);
    for (int n = 1; n < numberOfElements + 1; n++)
    {
        if (n % 2 == 0)
        {
            //Faza parzysta
            EvenPhase << <numBlocksEvenPhase, THREADS_PER_BLOCK >> > (d_newArray, numberOfElements);
        }
        else
        {
            //Faza nieparzysta 
            OddPhase << <numBlocksOddPhase, THREADS_PER_BLOCK >> > (d_newArray, numberOfElements);
        }
    }
    hipMemcpy(newArray, d_newArray, array_size, hipMemcpyDeviceToHost);
    stop = clock();

    if (time != NULL)
        *time = ((double)(stop - start)) / CLOCKS_PER_SEC;

    //Czyszczenie pamieci na device
    hipFree(d_newArray);

    return newArray;
}


int GetRandomInt(int lowerBound, int upperBound)
{
    return (rand() % (upperBound - lowerBound)) + lowerBound;
}


int * GenerateEvenSizeRandomElementArray(int seed, int* numberOfElements,
    int lowerBound, int upperBound)
{

    if (lowerBound >= upperBound)
    {
        printf("Warning! Wrong bounds, empty Array returned.\n");
        int * array;
        array = NULL;
        return array;
    }

    //Ustawiamy seed RNG - funkcja rand() za ka�dym razem zwr�ci ten sam 
    //ci�g liczb
    srand(seed);

    //Sprawdzamy czy liczba element�w ustawiona jest na parzyst�
    int makeEven = 0;
    if (*numberOfElements % 2 == 1)
    {
        printf("Warning! Generating array of uneven size, added additional element.\n");
        //Ustawiamy zmienn� pomocnicz�, kt�ra pozwoli 
        //wygenerowa� dodatkowy element
        makeEven = 1;
    }

    int * array;
    *numberOfElements = *numberOfElements + makeEven;
    array = (int*)malloc((*numberOfElements) * sizeof(int));
    //Losujemy parzyst� liczb� element�w
    for (size_t i = 0; i < *numberOfElements; i++)
    {
        array[i] = GetRandomInt(lowerBound, upperBound);
    }

    return array;
}


int * CopyArray(int * array, int numberOfElements)
{
    int * newArray;
    //Alokacja pami�ci
    newArray = (int*)malloc((numberOfElements) * sizeof(int));

    //Przepisywanie warto�ci element�w
    for (int i = 0; i < numberOfElements; i++)
    {
        newArray[i] = array[i];
    }

    return newArray;
}


void PrintElementArray(int * array, int numberOfElements)
{
    //Drukowanie warto�ci element�w
    for (size_t i = 0; i < numberOfElements; i++)
    {
        printf("%d ", array[i]);
    }
    printf("\n");
}



int main(int argc, char* argv[])
{
    int maxNumberOfElements = N;

	//Generowanie losowej tablicy o parzystej wielko�ci
	int* array;
	array = GenerateEvenSizeRandomElementArray(SEED, &maxNumberOfElements, 0, 100);
	printf("Input array size: %d\n\n", maxNumberOfElements);

	//Sortowanie
	double time;
	printf("CUDA version. Sorting...\n");
	int* sortedArray = TranspositionSort(array, maxNumberOfElements, &time);

	//Wypisywanie czasu
	printf("Sorting time (CUDA): %lf s\n\n", time);

	//Wy�wietlanie ma�ych tablic
	if (maxNumberOfElements <= 30)
	{
		printf("Array:\n");
		PrintElementArray(array, maxNumberOfElements);
		printf("Sorted array (CUDA):\n");
		PrintElementArray(sortedArray, maxNumberOfElements);
	}
    //Zwolnienie pamieci
	free(array);
	free(sortedArray);

    exit(EXIT_SUCCESS);
}